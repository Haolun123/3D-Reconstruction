#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <vector>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <fstream>
#include <math.h>
#define pi 3.1415926535
using namespace cv;
using std::cout;
using std::endl;


__global__ void binarykernel(uchar *dinput1,uchar *dinput2, uchar *dinput3, uchar *dinput4, uchar *dinput5,double doutbinary1[][1024],double doutbinary2[][1024],double doutbinary3[][1024],double doutbinary4[][1024],double doutbinary5[][1024] )
{
	int labelx = (blockIdx.x * blockDim.x + threadIdx.x);
	int labely = (blockIdx.y * blockDim.y + threadIdx.y);
	double p1 = dinput1[1280*labely+labelx];
	doutbinary1[labelx][labely]=(p1 > 255*0.3) ? 1 : 0;
	double p2 = dinput2[1280*labely+labelx];
	doutbinary2[labelx][labely]=(p2> 255*0.3) ? 1 : 0;
	double p3 = dinput3[1280*labely+labelx];
	doutbinary3[labelx][labely]=(p3 > 255*0.3) ? 1 : 0;
	double p4 = dinput4[1280*labely+labelx];
	doutbinary4[labelx][labely]=(p4 > 255*0.3) ? 1 : 0;
	double p5 = dinput5[1280*labely+labelx];
	doutbinary5[labelx][labely]=(p5 > 255*0.3) ? 1 : 0;
}
__global__ void phasewrapkernel(uchar *dph1, uchar *dph2, uchar *dph3, uchar *dph4, double dphasewrap[][1024])
{
	int labelx = (blockIdx.x * blockDim.x + threadIdx.x);
	int labely = (blockIdx.y * blockDim.y + threadIdx.y);
	double doutputsin = 0;
	double doutputcos = 0;
	dphasewrap[labelx][labely] = 0;
	double p1 = dph1[1280*labely+labelx];
	doutputsin += p1*sin(pi/2);
	doutputcos += p1*cos(pi/2);
	double p2 = dph2[1280*labely+labelx];
	doutputsin += p2*sin(2*pi/2);
	doutputcos += p2*cos(2*pi/2);
	double p3 = dph3[1280*labely+labelx];
	doutputsin += p3*sin(3*pi/2);
	doutputcos += p3*cos(3*pi/2);
	double p4 = dph4[1280*labely+labelx];
	doutputsin += p4*sin(4*pi/2);
	doutputcos += p4*cos(4*pi/2);
	dphasewrap[labelx][labely] = atan2(doutputsin,doutputcos);
}
__global__ void graykernel(double dG1[][1024],double dG2[][1024],double dG3[][1024],double dG4[][1024],double dG5[][1024],double dgraycode[][1024])
{
	int labelx = (blockIdx.x * blockDim.x + threadIdx.x);
	int labely = (blockIdx.y * blockDim.y + threadIdx.y);
	int gray1 = dG1[labelx][labely];
	int gray2 = dG2[labelx][labely];
	int gray3 = dG3[labelx][labely];
	int gray4 = dG4[labelx][labely];
	int gray5 = dG5[labelx][labely];
	dgraycode[labelx][labely] = gray1*16+(gray1^gray2)*8+((gray1^gray2)^gray3)*4+(((gray1^gray2)^gray3)^gray4)*2+(((gray1^gray2)^gray3)^gray4)^gray5;
}
 __global__ void constructimgkernel(double doutbinary1[][1024],double doutbinary2[][1024],double doutbinary3[][1024],double doutbinary4[][1024],double doutbinary5[][1024] )
{
	int labelx = (blockIdx.x * blockDim.x + threadIdx.x);
	int labely = (blockIdx.y * blockDim.y + threadIdx.y);
	double objphase = doutbinary1[labelx][labely];
	double objgray = doutbinary2[labelx][labely];
	double platephase = doutbinary3[labelx][labely];
	double plategray = doutbinary4[labelx][labely];
	doutbinary5[labelx][labely] = objphase+objgray*2*pi-platephase-plategray*2*pi;
}
__global__ void MedianFilter(double In[][1024],double Out[][1024])  
{  
    double window[9];  
    int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);  
    if(x>= 1280 && y>= 1024) return;  
    window[0]=(y==0||x==0)?0:In[x-1][y-1];  
    window[1]=(y==0)?0:In[x][y-1];  
    window[2]=(y==0||x==1279)? 0:In[x+1][y-1];  
    window[3]=(x==0)? 0:In[x-1][y];  
    window[4]= In[x][y];  
    window[5]=(x==1279)? 0:In[x+1][y];  
    window[6]=(y==1023||x==0)? 0:In[x-1][y+1];  
    window[7]=(y==1023)? 0:In[x][y+1];  
    window[8]=(y==1023||x==1279)? 0:In[x+1][y+1];  
    for (unsigned int j=0; j<5; ++j)  
    {  
        int min=j;  
        for (unsigned int l=j+1; l<9; ++l)  
            if (window[l] < window[min])  
                min=l;  
        double temp=window[j];  
        window[j]=window[min];  
        window[min]=temp;  
    }  
    Out[x][y]=window[4];  
}
int main()
{
	int i=0,j=0;
	Mat *objectgray=new Mat[5];
	Mat *plategray=new Mat[5];
	Mat *objectphase=new Mat[4];
	Mat *platephase=new Mat[4];
	dim3 grid( 64, 64 ), threads( 20, 16 );
	
	for(i=0;i<5;i++){  
		objectgray[i] = imread( format( "Capture%d.bmp",i+1),0); 
		plategray[i] = imread( format( "grayplate%d.bmp",i+1),0);
	}
	uchar *objectgray1 = objectgray[0].data;
	uchar *objectgray2 = objectgray[1].data;
	uchar *objectgray3 = objectgray[2].data;
	uchar *objectgray4 = objectgray[3].data;
	uchar *objectgray5 = objectgray[4].data;
	uchar *plategray1 = plategray[0].data;
	uchar *plategray2 = plategray[1].data;
	uchar *plategray3 = plategray[2].data;
	uchar *plategray4 = plategray[3].data;
	uchar *plategray5 = plategray[4].data;
	double (*objectbw1)[1024] = new double[1280][1024];
	double (*objectbw2)[1024] = new double[1280][1024];
	double (*objectbw3)[1024] = new double[1280][1024];
	double (*objectbw4)[1024] = new double[1280][1024];
	double (*objectbw5)[1024] = new double[1280][1024];
	double (*platebw1)[1024] = new double[1280][1024];
	double (*platebw2)[1024] = new double[1280][1024];
	double (*platebw3)[1024] = new double[1280][1024];
	double (*platebw4)[1024] = new double[1280][1024];
	double (*platebw5)[1024] = new double[1280][1024];
	double (*graycodeobject)[1024] = new double[1280][1024];
	double (*graycodeplate)[1024] = new double[1280][1024];
	uchar *dinput1,*dinput2,*dinput3,*dinput4,*dinput5;
	double (*doutbinary1)[1024],(*doutbinary2)[1024],(*doutbinary3)[1024],(*doutbinary4)[1024],(*doutbinary5)[1024];
	double (*dgraycode)[1024];
	hipMalloc((void**)&dinput1,1280*1024*sizeof(uchar));
	hipMalloc((void**)&dinput2,1280*1024*sizeof(uchar));
	hipMalloc((void**)&dinput3,1280*1024*sizeof(uchar));
	hipMalloc((void**)&dinput4,1280*1024*sizeof(uchar));
	hipMalloc((void**)&dinput5,1280*1024*sizeof(uchar));
	hipMalloc((void**)&doutbinary1,1280*1024*sizeof(double));
	hipMalloc((void**)&doutbinary2,1280*1024*sizeof(double));
	hipMalloc((void**)&doutbinary3,1280*1024*sizeof(double));
	hipMalloc((void**)&doutbinary4,1280*1024*sizeof(double));
	hipMalloc((void**)&doutbinary5,1280*1024*sizeof(double));
	hipMalloc((void**)&dgraycode,1280*1024*sizeof(double));
	hipMemcpy( dinput1, objectgray1, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dinput2, objectgray2, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dinput3, objectgray3, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dinput4, objectgray4, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dinput5, objectgray5, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	binarykernel<<<grid, threads>>>(dinput1,dinput2,dinput3,dinput4,dinput5,doutbinary1,doutbinary2,doutbinary3,doutbinary4,doutbinary5);
	graykernel<<<grid, threads>>>(doutbinary1,doutbinary2,doutbinary3,doutbinary4,doutbinary5,dgraycode);
	hipMemcpy( graycodeobject, dgraycode, 1280*1024*sizeof(double), hipMemcpyDeviceToHost ) ;
	
	hipMemcpy( dinput1, plategray1, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dinput2, plategray2, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dinput3, plategray3, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dinput4, plategray4, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dinput5, plategray5, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	binarykernel<<<grid, threads>>>(dinput1,dinput2,dinput3,dinput4,dinput5,doutbinary1,doutbinary2,doutbinary3,doutbinary4,doutbinary5);
	graykernel<<<grid, threads>>>(doutbinary1,doutbinary2,doutbinary3,doutbinary4,doutbinary5,dgraycode);
	hipMemcpy( graycodeplate, dgraycode, 1280*1024*sizeof(double), hipMemcpyDeviceToHost  );
	hipFree(dinput1);
	hipFree(dinput2);
	hipFree(dinput3);
	hipFree(dinput4);
	hipFree(dinput5);
	hipFree(doutbinary1);
	hipFree(doutbinary2);
	hipFree(doutbinary3);
	hipFree(doutbinary4);
	hipFree(doutbinary5);
	hipFree(dgraycode);
	for(j=0;j<4;j++){
		objectphase[j] = imread( format( "Capture0%d.bmp",j+1),0); 
		platephase[j] = imread( format( "plate%d.bmp",j+1),0);
	}
	uchar *objphase1 = objectphase[0].data;
	uchar *objphase2 = objectphase[1].data;
	uchar *objphase3 = objectphase[2].data;
	uchar *objphase4 = objectphase[3].data;
	uchar *platephase1 = platephase[0].data;
	uchar *platephase2 = platephase[1].data;
	uchar *platephase3 = platephase[2].data;
	uchar *platephase4 = platephase[3].data;
	double (*objphasewrap)[1024] = new double[1280][1024];
	double (*platephasewrap)[1024] = new double[1280][1024];
	uchar *dph1,*dph2,*dph3,*dph4;
	double (*dphasewrap)[1024];
	hipMalloc((void**)&dph1,1280*1024*sizeof(uchar));
	hipMalloc((void**)&dph2,1280*1024*sizeof(uchar));
	hipMalloc((void**)&dph3,1280*1024*sizeof(uchar));
	hipMalloc((void**)&dph4,1280*1024*sizeof(uchar));
	hipMalloc((void**)&dphasewrap,1280*1024*sizeof(double));
	hipMemcpy( dph1, objphase1, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dph2, objphase2, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dph3, objphase3, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dph4, objphase4, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	phasewrapkernel<<<grid, threads>>>(dph1,dph2,dph3,dph4,dphasewrap);
	hipMemcpy( objphasewrap, dphasewrap, 1280*1024*sizeof(double), hipMemcpyDeviceToHost ) ;

	hipMemcpy( dph1, platephase1, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dph2, platephase2, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dph3, platephase3, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	hipMemcpy( dph4, platephase4, 1280*1024*sizeof(uchar), hipMemcpyHostToDevice ) ;
	phasewrapkernel<<<grid, threads>>>(dph1,dph2,dph3,dph4,dphasewrap);
	hipMemcpy( platephasewrap, dphasewrap, 1280*1024*sizeof(double), hipMemcpyDeviceToHost ) ;
	hipFree(dph1);
	hipFree(dph2);
	hipFree(dph3);
	hipFree(dph4);
	hipFree(dphasewrap);
		
	for (i=0;i<1280;i++){
		for (j=0;j<1023;j++){
			if ((graycodeobject[i][j]==graycodeobject[i][j+1])&&(objphasewrap[i][j+1]-objphasewrap[i][j]>=pi))
				graycodeobject[i][j+1]=graycodeobject[i][j+1]-1;
			else if ((graycodeobject[i][j]==graycodeobject[i][j+1]+1)&&(objphasewrap[i][j+1]-objphasewrap[i][j]<pi))
					graycodeobject[i][j+1]=graycodeobject[i][j+1]+1;
				else if (graycodeobject[i][j]==graycodeobject[i][j+1]-1)
						graycodeobject[i][j+1]=graycodeobject[i][j];
		}
	}
	for (i=0;i<1280;i++){
		for (j=0;j<1023;j++){
			if ((graycodeplate[i][j]==graycodeplate[i][j+1])&&(platephasewrap[i][j+1]-platephasewrap[i][j]>=pi))
				graycodeplate[i][j+1]=graycodeplate[i][j+1]-1;
			else if ((graycodeplate[i][j]==graycodeplate[i][j+1]+1)&&(platephasewrap[i][j+1]-platephasewrap[i][j]<pi))
					graycodeplate[i][j+1]=graycodeplate[i][j+1]+1;
				else if (graycodeplate[i][j]==graycodeplate[i][j+1]-1)
						graycodeplate[i][j+1]=graycodeplate[i][j];
		}
	}	
	double (*imgoutput)[1024] = new double[1280][1024];
	double (*imgafterfilter)[1024] = new double[1280][1024];
	hipMalloc((void**)&doutbinary1,1280*1024*sizeof(double));
	hipMalloc((void**)&doutbinary2,1280*1024*sizeof(double));
	hipMalloc((void**)&doutbinary3,1280*1024*sizeof(double));
	hipMalloc((void**)&doutbinary4,1280*1024*sizeof(double));
	hipMalloc((void**)&doutbinary5,1280*1024*sizeof(double));
	hipMemcpy( doutbinary1, objphasewrap,  1280*1024*sizeof(double), hipMemcpyHostToDevice ) ;
	hipMemcpy( doutbinary2, graycodeobject,  1280*1024*sizeof(double), hipMemcpyHostToDevice ) ;
	hipMemcpy( doutbinary3, platephasewrap,  1280*1024*sizeof(double), hipMemcpyHostToDevice ) ;
	hipMemcpy( doutbinary4, graycodeplate,  1280*1024*sizeof(double), hipMemcpyHostToDevice ) ;
	constructimgkernel<<<grid, threads>>>(doutbinary1,doutbinary2,doutbinary3,doutbinary4,doutbinary5);
	hipMemcpy( imgoutput, doutbinary5,  1280*1024*sizeof(double), hipMemcpyDeviceToHost ) ;
	hipFree(doutbinary2);
	hipFree(doutbinary3);
	hipFree(doutbinary4);
	MedianFilter<<<grid, threads>>>(doutbinary5,doutbinary1);
	hipMemcpy( imgafterfilter, doutbinary1,  1280*1024*sizeof(double), hipMemcpyDeviceToHost ) ;
	hipFree(doutbinary1);
	hipFree(doutbinary5);
	std::ofstream outf("out.txt",std::ios::out);
   
	for (i=300;i<700;i++){
		for (j=400;j<900;j++){
			double z=imgafterfilter[j][i];
			outf<<j-400<<" "<<i-300<<" "<<z<<endl;
		}
	}
	outf.close();
	
}

